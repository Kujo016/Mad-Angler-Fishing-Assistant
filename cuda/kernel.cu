#include "hip/hip_runtime.h"
﻿#include "../cuda/kernel.cuh"





// ✅ GPU Kernel: Checks if a keyword exists in each line
__device__ bool contains(const char* line, const char* keyword) {
    int i = 0;
    while (line[i] != '\0') {
        int j = 0;
        while (line[i + j] != '\0' && keyword[j] != '\0' && line[i + j] == keyword[j]) {
            j++;
        }
        if (keyword[j] == '\0') return true;
        i++;
    }
    return false;
}

// ✅ CUDA Kernel for text tagging
__global__ void tag_text_lines(char* lines, char* keywords, int* results, int num_lines, int num_keywords, int max_line_length) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < num_lines) {
        char* line = lines + idx * max_line_length;

        for (int k = 0; k < num_keywords; k++) {
            char* keyword = keywords + k * 32;  // Max keyword length of 32
            if (contains(line, keyword)) {
                results[idx] = k + 1;  // Store keyword index if found
            }
        }
    }
}

// ✅ Read a text file and return its lines as a vector
std::vector<std::string> read_txt(const std::string& filepath) {
    std::vector<std::string> lines;
    std::ifstream file(filepath, std::ios::binary); // Open in binary mode to avoid newline conversion

    if (!file) {
        std::cerr << "[ERROR] Cannot open file: " << filepath << std::endl;
        return lines; // Return empty vector
    }

    std::string line;
    while (std::getline(file, line)) {
        // ✅ Remove invalid UTF-8 characters
        std::string sanitizedLine = removeInvalidUtf8(line);

        // ✅ Ignore completely empty lines
        if (!sanitizedLine.empty()) {
            lines.push_back(sanitizedLine);
        }
    }

    file.close();
    return lines;
}

// ✅ Remove invalid UTF-8 sequences from a string
std::string removeInvalidUtf8(const std::string& input) {
    std::string output;
    int i = 0;
    while (i < input.size()) {
        unsigned char c = static_cast<unsigned char>(input[i]);
        if (c < 0x80) {  // ASCII characters (valid)
            output += c;
            i++;
        }
        else if ((c & 0xE0) == 0xC0 && i + 1 < input.size() &&
            (static_cast<unsigned char>(input[i + 1]) & 0xC0) == 0x80) {
            // ✅ Valid 2-byte UTF-8 sequence
            output += input[i];
            output += input[i + 1];
            i += 2;
        }
        else if ((c & 0xF0) == 0xE0 && i + 2 < input.size() &&
            (static_cast<unsigned char>(input[i + 1]) & 0xC0) == 0x80 &&
            (static_cast<unsigned char>(input[i + 2]) & 0xC0) == 0x80) {
            // ✅ Valid 3-byte UTF-8 sequence
            output += input[i];
            output += input[i + 1];
            output += input[i + 2];
            i += 3;
        }
        else if ((c & 0xF8) == 0xF0 && i + 3 < input.size() &&
            (static_cast<unsigned char>(input[i + 1]) & 0xC0) == 0x80 &&
            (static_cast<unsigned char>(input[i + 2]) & 0xC0) == 0x80 &&
            (static_cast<unsigned char>(input[i + 3]) & 0xC0) == 0x80) {
            // ✅ Valid 4-byte UTF-8 sequence
            output += input[i];
            output += input[i + 1];
            output += input[i + 2];
            output += input[i + 3];
            i += 4;
        }
        else {
            // ❌ Invalid UTF-8 character: Skip it
            std::cerr << "[WARNING] Removed invalid UTF-8 byte: " << std::hex << static_cast<int>(c) << std::dec << std::endl;
            i++;
        }
    }
    return output;
}

// ✅ Process text files with CUDA
__host__ json process_text_files(const std::string& filepath, const std::unordered_map<std::string, std::vector<std::string>>& tags) {
    std::vector<std::string> lines = read_txt(filepath);
    int num_lines = static_cast<int>(lines.size());
    if (num_lines == 0 || tags.empty()) return json();

    // Flatten the tag list into keyword and category arrays
    std::vector<std::string> keyword_list, category_list;
    for (const auto& kv : tags) {
        const std::string& category = kv.first;
        for (const std::string& keyword : kv.second) {
            std::string lower_keyword = keyword;
            std::transform(lower_keyword.begin(), lower_keyword.end(), lower_keyword.begin(), ::tolower);
            keyword_list.push_back(lower_keyword);
            category_list.push_back(category);
        }
    }

    int num_keywords = static_cast<int>(keyword_list.size());
    if (num_keywords == 0) return json();

    // ✅ Allocate CPU memory
    char* h_lines = new char[num_lines * MAX_LINE_LENGTH]();
    char* h_keywords = new char[num_keywords * 32]();
    int* h_results = new int[num_lines]();

    // ✅ Copy lines & keywords to buffers
    for (int i = 0; i < num_lines; i++) {
        std::string lower_line = lines[i];
        std::transform(lower_line.begin(), lower_line.end(), lower_line.begin(), ::tolower);
        strncpy(h_lines + i * MAX_LINE_LENGTH, lower_line.c_str(), MAX_LINE_LENGTH - 1);
    }
    for (int i = 0; i < num_keywords; i++) {
        strncpy(h_keywords + i * 32, keyword_list[i].c_str(), 31);
        h_keywords[i * 32 + 31] = '\0';
    }

    // ✅ Allocate GPU memory
    char* d_lines, * d_keywords;
    int* d_results;
    hipMalloc(&d_lines, num_lines * MAX_LINE_LENGTH);
    hipMalloc(&d_keywords, num_keywords * 32);
    hipMalloc(&d_results, num_lines * sizeof(int));
    hipMemset(d_results, 0, num_lines * sizeof(int));


    hipMemcpy(d_lines, h_lines, num_lines * MAX_LINE_LENGTH, hipMemcpyHostToDevice);
    hipMemcpy(d_keywords, h_keywords, num_keywords * 32, hipMemcpyHostToDevice);
    hipMemcpy(d_results, h_results, num_lines * sizeof(int), hipMemcpyHostToDevice);

    // ✅ Define CUDA launch parameters
    int threadsPerBlock = 256;
    int blocksPerGrid = (num_lines + threadsPerBlock - 1) / threadsPerBlock;

    // ✅ Launch CUDA Kernel
    tag_text_lines << <blocksPerGrid, threadsPerBlock >> > (d_lines, d_keywords, d_results, num_lines, num_keywords, MAX_LINE_LENGTH);
    hipDeviceSynchronize();

    // ✅ Copy results back from GPU
    hipMemcpy(h_results, d_results, num_lines * sizeof(int), hipMemcpyDeviceToHost);

    // ✅ Populate JSON output
    json output, summary;
    bool foundMatch = false;
    for (int i = 0; i < num_lines; i++) {
        std::string sanitizedLine = removeInvalidUtf8(lines[i]);
        // Check for a valid match: > 0 and within range
        if (h_results[i] > 0 && h_results[i] <= num_keywords) {
            foundMatch = true;
            // Adjust the index by subtracting 1 since we stored k+1
            std::string matched_category = category_list[h_results[i] - 1];
            summary[matched_category].push_back(sanitizedLine);
        }
    }
    output["summary"] = foundMatch ? summary : json({ {"error", "No matches found"} });


    // ✅ Free memory
    hipFree(d_lines);
    hipFree(d_keywords);
    hipFree(d_results);
    delete[] h_lines;
    delete[] h_keywords;
    delete[] h_results;

    return output;
}

// ✅ CUDA Kernel to extract keyword matches from text data
__global__ void extract_tags_kernel(char* fileData, int fileSize, char* tags, int* tagOffsets, int numTags, int* results) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= fileSize) return; // Ensure within file bounds

    for (int t = 0; t < numTags; t++) {
        // Use provided tagOffsets if available, else compute offset as t * MAX_KEYWORD_LENGTH
        int tagStart = (tagOffsets != nullptr) ? tagOffsets[t] : t * MAX_KEYWORD_LENGTH;
        char* keyword = &tags[tagStart];

        // Check if keyword is found at this position
        int j = 0;
        while (keyword[j] != '\0' && fileData[idx + j] == keyword[j]) {
            j++;
        }

        if (keyword[j] == '\0') {
            // Mark match in results array
            atomicAdd(&results[t], 1);
        }
    }
}

// ✅ Extract Tags from CUDA processing
void load_tags_cuda(const std::string& filepath, std::unordered_set<std::string>& tags) {
    std::ifstream file(filepath, std::ios::binary | std::ios::ate);
    if (!file) {
        std::cerr << "[ERROR] Cannot open file: " << filepath << std::endl;
        return;
    }

    size_t fileSize = file.tellg();
    file.seekg(0);
    std::vector<char> fileData(fileSize);
    file.read(fileData.data(), fileSize);
    file.close();

    char* d_fileData;
    hipMalloc((void**)&d_fileData, fileSize);
    hipMemcpy(d_fileData, fileData.data(), fileSize, hipMemcpyHostToDevice);

    int numTags = 0;
    int* d_numTags;
    hipMalloc((void**)&d_numTags, sizeof(int));
    hipMemcpy(d_numTags, &numTags, sizeof(int), hipMemcpyHostToDevice);

    // ✅ Launch Kernel
    int numBlocks = (fileSize + BLOCK_SIZE - 1) / BLOCK_SIZE;
    extract_tags_kernel << <numBlocks, BLOCK_SIZE >> > (d_fileData, fileSize, nullptr, nullptr, numTags, d_numTags);
    hipMemcpy(&numTags, d_numTags, sizeof(int), hipMemcpyDeviceToHost);

    // ✅ Free memory
    hipFree(d_fileData);
    hipFree(d_numTags);
}

// ✅ Process text with CUDA
void process_text_with_cuda(std::string& text, std::vector<std::string>& tags, std::vector<int>& results) {
    int text_size = text.size();
    int num_tags = tags.size();

    char* d_text, * d_tags;
    int* d_results;
    hipMalloc(&d_text, text_size * sizeof(char));
    hipMalloc(&d_tags, num_tags * MAX_KEYWORD_LENGTH * sizeof(char));
    hipMalloc(&d_results, num_tags * sizeof(int));

    hipMemcpy(d_text, text.c_str(), text_size * sizeof(char), hipMemcpyHostToDevice);

    std::vector<char> h_tags(num_tags * MAX_KEYWORD_LENGTH, 0);
    for (int i = 0; i < num_tags; i++) {
        strncpy(&h_tags[i * MAX_KEYWORD_LENGTH], tags[i].c_str(), MAX_KEYWORD_LENGTH);
    }

    hipMemcpy(d_tags, h_tags.data(), num_tags * MAX_KEYWORD_LENGTH * sizeof(char), hipMemcpyHostToDevice);
    hipMemset(d_results, 0, num_tags * sizeof(int));

    int numBlocks = (text_size + BLOCK_SIZE - 1) / BLOCK_SIZE;
    extract_tags_kernel << <numBlocks, BLOCK_SIZE >> > (d_text, text_size, d_tags, nullptr, num_tags, d_results);

    results.resize(num_tags);
    hipMemcpy(results.data(), d_results, num_tags * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_text);
    hipFree(d_tags);
    hipFree(d_results);
}

//Called From Python
// ----- CUDA Kernel -----
__global__ void normalize_weather_data(WeatherPoint* data, size_t count) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= count) return;

    data[i].temperature /= 100.0f;
    data[i].humidity /= 100.0f;
    data[i].pressure /= 1000.0f;
}

// ----- CUDA Processing -----
void process_weather_with_cuda(std::vector<WeatherPoint>& points) {
    if (points.empty()) return;

    WeatherPoint* d_data;
    size_t size = points.size() * sizeof(WeatherPoint);

    hipMalloc(&d_data, size);
    hipMemcpy(d_data, points.data(), size, hipMemcpyHostToDevice);

    int blockSize = 256;
    int gridSize = (points.size() + blockSize - 1) / blockSize;
    normalize_weather_data << <gridSize, blockSize >> > (d_data, points.size());

    hipDeviceSynchronize();
    hipMemcpy(points.data(), d_data, size, hipMemcpyDeviceToHost);
    hipFree(d_data);
}

// Helper to process a CSV line into float values
std::vector<float> parse_csv_line(const std::string& line) {
    std::vector<float> values;
    size_t start = 0;
    size_t end = line.find(',');

    while (end != std::string::npos) {
        values.push_back(std::stof(line.substr(start, end - start)));
        start = end + 1;
        end = line.find(',', start);
    }
    values.push_back(std::stof(line.substr(start)));

    return values;
}

__global__ void compute_trajectory(const WeatherPoint* input, TrajectoryPoint* output, int count) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= count - 1) return;

    float dx = input[i + 1].temperature - input[i].temperature;
    float dy = input[i + 1].humidity - input[i].humidity;
    float dz = input[i + 1].pressure - input[i].pressure;

    float mag = sqrtf(dx * dx + dy * dy + dz * dz);

    int quadrant = 0;
    if (dx > 0 && dy > 0) quadrant = 1;
    else if (dx < 0 && dy > 0) quadrant = 2;
    else if (dx < 0 && dy < 0) quadrant = 3;
    else if (dx > 0 && dy < 0) quadrant = 4;

    output[i] = { dx, dy, dz, quadrant, mag };
}

void compute_weather_trajectory(const std::vector<WeatherPoint>& input, std::vector<TrajectoryPoint>& output) {
	if (input.size() < 2) return;

    int count = static_cast<int>(input.size());
    size_t input_size = count * sizeof(WeatherPoint);
    size_t output_size = (count - 1) * sizeof(TrajectoryPoint);

    WeatherPoint* d_input = nullptr;
    TrajectoryPoint* d_output = nullptr;

    // Allocate GPU memory
    hipMalloc(&d_input, input_size);
    hipMalloc(&d_output, output_size);

    // Copy input to device
    hipMemcpy(d_input, input.data(), input_size, hipMemcpyHostToDevice);

    // Launch kernel
    int blockSize = 256;
    int gridSize = (count - 1 + blockSize - 1) / blockSize;
    compute_trajectory << <gridSize, blockSize >> > (d_input, d_output, count);

    // Wait for GPU to finish
    hipDeviceSynchronize();

    // Prepare output vector and copy back
    output.resize(count - 1);
    hipMemcpy(output.data(), d_output, output_size, hipMemcpyDeviceToHost);

    // Cleanup
    hipFree(d_input);
    hipFree(d_output);
}